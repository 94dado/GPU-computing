#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "common.h"


/*  Carve the maze starting at x, y. */
void CarveMaze(int *maze, int width, int height, int x, int y) {

   int x1, y1;
   int x2, y2;
   int dx, dy;
   int dir, count;

   dir = rand() % 4;
   count = 0;
   while(count < 4) {
      dx = 0; dy = 0;
      switch(dir) {
      case 0:  dx = 1;  break;
      case 1:  dy = 1;  break;
      case 2:  dx = -1; break;
      default: dy = -1; break;
      }
      x1 = x + dx;
      y1 = y + dy;
      x2 = x1 + dx;
      y2 = y1 + dy;
      if(   x2 > 0 && x2 < width && y2 > 0 && y2 < height
         && maze[y1 * width + x1] == WALL && maze[y2 * width + x2] == WALL) {
         maze[y1 * width + x1] = OPEN;
         maze[y2 * width + x2] = OPEN;
         x = x2; y = y2;
         dir = rand() % 4;
         count = 0;
      } else {
         dir = (dir + 1) % 4;
         count += 1;
      }
   }

}

/* Generate maze in matrix maze with size width, height. */
void CPU_backtracker_maze_generator(int *maze, int width, int height) {
   int x, y;

   /* Initialize the maze. */
   for(x = 0; x < width * height; x++) {
      maze[x] = WALL;
   }
   maze[1 * width + 1] = OPEN;

   /* Carve the maze. */
   for(y = 1; y < height; y += 2) {
      for(x = 1; x < width; x += 2) {
         CarveMaze(maze, width, height, x, y);
      }
   }

   /* Set up the entry and exit. */
   maze[0 * width + 1] = OPEN;
   maze[(height - 1) * width + (width - 2)] = OPEN;
}

//int main(){
//	int maze[100];
//	CPU_backtracker_maze_generator(maze,10,10);
//	print_maze(maze,10,10);
//	return 0;
//}


/* Solve the maze.
void SolveMaze(char *maze, int width, int height) {

   int dir, count;
   int x, y;
   int dx, dy;
   int forward;

   // Remove the entry and exit.
   maze[0 * width + 1] = 1;
   maze[(height - 1) * width + (width - 2)] = 1;

   forward = 1;
   dir = 0;
   count = 0;
   x = 1;
   y = 1;
   while(x != width - 2 || y != height - 2) {
      dx = 0; dy = 0;
      switch(dir) {
      case 0:  dx = 1;  break;
      case 1:  dy = 1;  break;
      case 2:  dx = -1; break;
      default: dy = -1; break;
      }
      if(   (forward  && maze[(y + dy) * width + (x + dx)] == 0)
         || (!forward && maze[(y + dy) * width + (x + dx)] == 2)) {
         maze[y * width + x] = forward ? 2 : 3;
         x += dx;
         y += dy;
         forward = 1;
         count = 0;
         dir = 0;
      } else {
         dir = (dir + 1) % 4;
         count += 1;
         if(count > 3) {
            forward = 0;
            count = 0;
         }
      }
   }

   // Replace the entry and exit.
   maze[(height - 2) * width + (width - 2)] = 2;
   maze[(height - 1) * width + (width - 2)] = 2;
}
*/
