#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "Header/common.h"

/*  Carve the maze starting at x, y. */
void CarveMaze(int *maze, int width, int height, int x, int y) {

   int x1, y1;
   int x2, y2;
   int dx, dy;
   int dir, count;

   dir = rand() % 4;
   count = 0;
   while(count < 4) {
      dx = 0; dy = 0;
      switch(dir) {
      case 0:  dx = 1;  break;
      case 1:  dy = 1;  break;
      case 2:  dx = -1; break;
      default: dy = -1; break;
      }
      x1 = x + dx;
      y1 = y + dy;
      x2 = x1 + dx;
      y2 = y1 + dy;
      if(   x2 > 0 && x2 < width && y2 > 0 && y2 < height
         && maze[y1 * width + x1] == WALL && maze[y2 * width + x2] == WALL) {
         maze[y1 * width + x1] = OPEN;
         maze[y2 * width + x2] = OPEN;
         x = x2; y = y2;
         dir = rand() % 4;
         count = 0;
      } else {
         dir = (dir + 1) % 4;
         count += 1;
      }
   }

}

/* Generate maze in matrix maze with size width, height. */
void CPU_backtracker_maze_generator(int *maze, int width, int height) {
   int x, y;

   srand(time(NULL));
   /* Initialize the maze. */
   for(x = 0; x < width * height; x++) {
      maze[x] = WALL;
   }
   maze[1 * width + 1] = OPEN;

   /* Carve the maze. */
   for(y = 1; y < height; y += 2) {
      for(x = 1; x < width; x += 2) {
         CarveMaze(maze, width, height, x, y);
      }
   }

   /* Set up the entry and exit. */
   maze[0 * width + 1] = OBJECTIVE;
   maze[(height - 1) * width + (width - 2)] = OBJECTIVE;
}

__global__ void GPU_init_maze(int *maze, int length, int row_length, int offset){
	int idx = blockIdx.x * row_length + offset + threadIdx.x;
	maze[idx] = WALL;
}

__global__ void GPU_carve_maze(int *maze,int width, int height, int rand1, int rand2, int i){
//	int y = blockIdx.x;
	int y = i;
	int x = threadIdx.y;
	//only odd numbers
	if(x%2 == 0 && y%2 == 0) return;

	int x1, y1;
	int x2, y2;
	int dx, dy;
	int dir, count;

	dir = rand1;
	count = 0;
	while(count < 4) {
	 dx = 0; dy = 0;
	 switch(dir) {
		 case 0:  dx = 1;  break;
		 case 1:  dy = 1;  break;
		 case 2:  dx = -1; break;
		 default: dy = -1; break;
	 }
	 x1 = x + dx;
	 y1 = y + dy;
	 x2 = x1 + dx;
	 y2 = y1 + dy;
	 if(x2 > 0 && x2 < width && y2 > 0 && y2 < height
		&& maze[y1 * width + x1] == WALL && maze[y2 * width + x2] == WALL) {
		 maze[y1 * width + x1] = OPEN;
		 maze[y2 * width + x2] = OPEN;
		 x = x2; y = y2;
		 dir = rand2;
		 count = 0;
	 } else {
		 dir = (dir + 1) % 4;
		 count += 1;
	 }
	}
}
void GPU_backtracker_maze_generator(int *maze, int width, int height){
	int *dev_maze;
	int length = width * height;
	srand(time(NULL));
	//initialize the maze
	hipMalloc(&dev_maze, sizeof(int) * length);
	hipMemcpy(dev_maze, maze, sizeof(int) * length, hipMemcpyHostToDevice);
	int max_rec = width / MAX_THREAD;
	int offset = 0;
	for(int i = 0; i < max_rec; i++){
		GPU_init_maze<<<height, MAX_THREAD>>>(dev_maze, width* height, width, offset);
		offset = (i+1) * MAX_THREAD;
	}
	GPU_init_maze<<<height, width % MAX_THREAD>>>(dev_maze, width* height, width, offset);
	hipDeviceSynchronize();
	hipMemcpy(maze,dev_maze, sizeof(int) * length, hipMemcpyDeviceToHost);

	maze[1 * width + 1] = OPEN;
	//carve the maze
	for(int y = 1; y < height; y += 2) {
	  for(int x = 1; x < width; x += 2) {
		 CarveMaze(maze, width, height, x, y);
	  }
	}

//	hipMemcpy(dev_maze, maze, sizeof(int) * length, hipMemcpyHostToDevice);
//	for(int i = 0; i < width; i++){	//per sicurezza lo lancio 2 volte
//		int rand1 = rand() % 4;
//		int rand2 = rand() % 4;
//		GPU_carve_maze<<<1, width>>>(dev_maze, width, height, rand1, rand2, i);
//		hipDeviceSynchronize();
//	}
//	hipMemcpy(maze,dev_maze, sizeof(int) * length, hipMemcpyDeviceToHost);

	/* Set up the entry and exit. */
	maze[0 * width + 1] = OBJECTIVE;
	maze[(height - 1) * width + (width - 2)] = OBJECTIVE;
}
