#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include "time.h"
#include "Header/common.h"

using namespace std;

#define HORIZONTAL 1
#define VERTICAL 2

#define MAZE_RESOLUTION 2


int choose_orientation(int width, int height){
	if(width < height)	return HORIZONTAL;
	else if(height < width) return VERTICAL;
	else return rand()%2 + 1;
}

void recursive_divide(int *maze, int x, int y, int width, int height, int orientation){
//	cout << "------------------" << endl << "recursion started" << endl;
//	cout << "x: " << x << ", y: " << y << ", width: " << width << ", height: " << height << ", orientation: " << orientation << endl;
	int fromX, fromY,holeX,holeY,directionX,directionY,wall_lenght;
	int newX, newY, newWidth, newHeight;
	//check if I don't have to make this iteration
	if(width < MAZE_RESOLUTION || height < MAZE_RESOLUTION){
		return;
	}
	if(orientation == HORIZONTAL){
		//is horizontal
		fromX = x;
		if(height - 2 <= 0){
			fromY = y;
		}else{
			fromY = y + rand()%(height-2);
		}
		holeX = fromX + rand()%width;
		holeY = fromY;
		directionX = 1;
		directionY = 0;
		wall_lenght = width;

	}else{
		//is vertical
		if(width - 2 <= 0){
			fromX = x;
		}else{
			fromX = x + rand()%(width-2);
		}
		fromY = y;
		holeX = fromX;
		holeY = fromY + rand()%height;
		directionX = 0;
		directionY = 1;
		wall_lenght = height;
	}
//	cout << "data calculated:" << endl;
//	cout << "fromX: " << fromX << ", fromY: " << fromY << ", holeX: " << holeX
//			<< ", holeY: " << holeY << ", directionX: " << directionX << ", wall_length:"
//			<< wall_lenght << ", perpendicular: " << perpendicular << endl;
	//create the wall with an hole
	for(int i = 0; i < wall_lenght; i++){
		if(fromX != holeX || fromY != holeY)
		maze[width * fromY + fromX] = WALL;
		fromX += directionX;
		fromY += directionY;
	}
	//first recursive call
	newX = x;
	newY = y;
	if(orientation == HORIZONTAL){
		newWidth = width;
		newHeight = fromY - y + 1;
	}else{
		newWidth = fromX - x + 1;
		newHeight = height;
	}
	recursive_divide(maze,newX, newY, newWidth, newHeight, choose_orientation(newWidth, newHeight));

	//second recursive call
	if(orientation == HORIZONTAL){
		newX = x;
		newY = fromY + 1;
		newWidth = width;
		newHeight = y + height - fromY - 1;
	}else{
		newX = fromX + 1;
		newY = y;
		newWidth = x + width - fromX - 1;
		newHeight = height;
	}
	recursive_divide(maze,newX, newY, newWidth, newHeight, choose_orientation(newWidth, newHeight));
}

void CPU_division_maze_generator(int *maze, int width, int height){
	//set random seed
	srand(time(NULL));
	//first, set everything to OPEN
	FillOpen(maze, width* height);
	//start with the algorithm
	recursive_divide(maze, 0, 0, width, height, choose_orientation(width, height));
}

__device__ int device_choose_orientation (int width, int height){
	if(width < height)	return HORIZONTAL;
	else if(height < width) return VERTICAL;
	else return 1;	//i removed random. Hope it doesn't suck at all
}

class StackElement {

	public:
		int x;
		int y;				//coords
		int width;
		int height;			//size of the sub-array
		int orientation;	//orientation for the algorithm
		int random_value;	//a random value

		//constructor
		StackElement(int _x, int _y, int _width, int _height){
			x = _x;
			y = _y;
			width = _width;
			height = _height;
			orientation = choose_orientation(width, height);
			random_value = -1;	//no setup for device compatibility code
		}

		__device__ StackElement(int _x, int _y, int _width, int _height, int useless){
			x = _x;
			y = _y;
			width = _width;
			height = _height;
			orientation = device_choose_orientation(width, height);
			random_value = -1;	//no setup for device compatibility code
		}


		__device__ StackElement(){
			x = y = width = height = orientation = random_value = -1;
		}

		__device__ bool isEmpty(){
			return x == y == width == height == orientation == -1;
		}

		__device__ void copyAttributes(int &_x, int &_y, int &_width, int &_height, int &_orientation, int &_rand){
			_x = x;
			_y = y;
			_width = width;
			_height = height;
			_orientation = orientation;
			_rand = random_value;

		}


};

//function that actually execute the code of the algorithm on the gpu
__global__ void GPU_iterator_divide(int *maze, StackElement *stack, int size_stack, StackElement *new_stack, int *size_new_stack, int *recursive_calls){
	//define parameters for original algorithm
	int x, y, width, height, orientation, rand;
	//get parameters values
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//if it's a valid index
	if(index < size_stack){
		//check if I have data
		if(stack[index].isEmpty()) return;
		//get data from the stack
		stack[index].copyAttributes(x,y,width,height,orientation,rand);
		//start algorithm
		int fromX, fromY,holeX, holeY, directionX, directionY, wall_lenght;
		int newX, newY, newWidth, newHeight;
		//check if I don't have to make this iteration
		if(width < MAZE_RESOLUTION || height < MAZE_RESOLUTION){
			new_stack[2 * index] = StackElement();
			new_stack[2* index + 1] = StackElement();
			atomicAdd(size_new_stack, 2);
			return;
		}
		if(orientation == HORIZONTAL){
			//is horizontal
			fromX = x;
			if(height - 2 <= 0){
				fromY = y;
			}else{
				fromY = y + rand %(height-2);
			}
			holeX = fromX + rand %width;
			holeY = fromY;
			directionX = 1;
			directionY = 0;
			wall_lenght = width;

		}else{
			//is vertical
			if(width - 2 <= 0){
				fromX = x;
			}else{
				fromX = x + rand %(width-2);
			}
			fromY = y;
			holeX = fromX;
			holeY = fromY + rand %height;
			directionX = 0;
			directionY = 1;
			wall_lenght = height;
		}
		//create the wall with an hole
		for(int i = 0; i < wall_lenght; i++){
			if(fromX != holeX || fromY != holeY)
			maze[width * fromY + fromX] = WALL;
			fromX += directionX;
			fromY += directionY;
		}
		//first recursive call
		newX = x;
		newY = y;
		if(orientation == HORIZONTAL){
			newWidth = width;
			newHeight = fromY - y + 1;
		}else{
			newWidth = fromX - x + 1;
			newHeight = height;
		}
		//add call in stack
		new_stack[index * 2] = StackElement(newX, newY, newWidth, newHeight, 0);
		atomicAdd(size_new_stack, 1);

		//second recursive call
		if(orientation == HORIZONTAL){
			newX = x;
			newY = fromY + 1;
			newWidth = width;
			newHeight = y + height - fromY - 1;
		}else{
			newX = fromX + 1;
			newY = y;
			newWidth = x + width - fromX - 1;
			newHeight = height;
		}
		//add call in stack
		new_stack[2* index + 1] = StackElement(newX, newY, newWidth, newHeight, NULL);
		atomicAdd(size_new_stack,1);

		//increment counter of calls
		atomicAdd(recursive_calls, 2);
	}
}

//function that iterate the call of the algorithm in GPU version
void GPU_recursive_divide(int *maze,int width, int height){
	int size_stack = 0;
	int *dev_size_temp;
	int recursive_calls = 0;
	int *dev_recursive;
	int default_size = 0;
	//create stack
	int stack_dimension = width * height * 100;
	StackElement stack[stack_dimension];
	//stack for device
	StackElement *dev_stack, *dev_temp_stack;
	hipMalloc(&dev_temp_stack, sizeof(StackElement) * stack_dimension);
	hipMalloc(&dev_size_temp, sizeof(int));
	hipMalloc(&dev_recursive, sizeof(int));

	bool again = true;
	//setup the first call
	stack[size_stack++] = StackElement(0, 0, width, height);
	while(again){	//till the end of the recursive calls
//		cout << "size_stack: " << size_stack << endl;
		//setup a random value for each stack call
		for(int i = 0; i < size_stack; i++){
			stack[i].random_value = rand();
		}
		int n_blocks = (size_stack / 32 ) + 1;
		//allocate and copy device stack
		hipMalloc(&dev_stack, sizeof(StackElement) * size_stack);
		hipMemcpy(dev_stack, stack, sizeof(StackElement) * size_stack, hipMemcpyHostToDevice);
		//setup size of the new stack and recursive calls
		hipMemcpy(dev_size_temp, &default_size, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_recursive, &default_size, sizeof(int), hipMemcpyHostToDevice);
		//call the real algorithm
//		cout << "calling gpu ... ";
		GPU_iterator_divide<<<n_blocks, 32>>>(maze, dev_stack, size_stack, dev_temp_stack, dev_size_temp, dev_recursive);
//		cout << "end!" << endl;
		//wait that every thread finishes
		hipDeviceSynchronize();
		//copy stack size and stack to host
		hipMemcpy(&size_stack, dev_size_temp, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(stack, dev_temp_stack, sizeof(StackElement) * size_stack, hipMemcpyDeviceToHost);
		hipMemcpy(&recursive_calls, dev_recursive, sizeof(int), hipMemcpyDeviceToHost);
		//free dev_stack
		hipFree(dev_stack);
		//check if I have finished
		if(recursive_calls == 0) again = false;
	}
}


void GPU_division_maze_generator(int *maze, int width, int height){
	//define maze on GPU
	int *dev_maze;
	hipMalloc(&dev_maze, sizeof(int) * width * height);
	hipMemcpy(dev_maze, maze, sizeof(int) * width * height, hipMemcpyHostToDevice);
	//set random seed
	srand(time(NULL));
	//first, set everything to OPEN
	GPU_FillOpen<<<height, width>>>(dev_maze, width, width* height);
	//start with the algorithm
	GPU_recursive_divide(dev_maze, width, height);
	//maze generated. copy on host
	hipMemcpy(maze, dev_maze, sizeof(int) * width * height, hipMemcpyDeviceToHost);
}

