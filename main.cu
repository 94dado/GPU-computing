#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include "Header/CPU_time.h"

#include "Header/common.h"
#include "Header/backtracker_maze_generator.h"
#include "Header/cellular_automata_solver.h"
#include "Header/dfs_maze_generator.h"
#include "Header/wall_follower_maze_solver.h"
#include "Header/division_maze_generator.h"
#include "Header/kruskal_maze_generator.h"
#include "Header/bfs_maze_solver.h"
#include "Header/recursive_maze_solver.h"

#define NUMBER_OF_TEST 1

//DFS: print with side-1

void dfs(int side, int *maze1, int *maze2) {
	double diff;
	cout << endl << "DFS MAZE GENERATOR WITH CPU" << endl;
	double start = seconds();
	CPU_dfs_maze_generator(maze1, side, side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze1,side-1,side-1);
	cout << endl << "DFS MAZE GENERATOR WITH GPU" << endl;
	start = seconds();
	GPU_dfs_maze_generator(maze2,side,side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze2,side-1,side-1);
	hipDeviceReset();
}

void cellular(int side, int *maze1, int *maze2) {
	double diff;
	cout << endl << "CELLULAR AUTOMATA MAZE SOLVER WITH CPU" << endl;
	double start = seconds();
	CPU_cellular_automata_solver(maze1, side, side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze1,side,side);
	cout << endl << "CELLULAR AUTOMATA MAZE SOLVER WITH GPU" << endl;
	start = seconds();
	GPU_cellular_automata_solver(maze2,side,side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze2,side,side);
	hipDeviceReset();
}

void backtracker(int side, int *maze1, int *maze2) {
	double diff;
	cout << endl << "BACKTRACKER MAZE GENERATOR WITH CPU" << endl;
	double start = seconds();
	CPU_backtracker_maze_generator(maze1, side, side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze1,side,side);
	cout << endl << "BACKTRACKER MAZE GENERATOR WITH GPU" << endl;
	start = seconds();
	GPU_backtracker_maze_generator(maze2,side,side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze2,side,side);
	hipDeviceReset();
}

void bfs(int side, int *maze1, int *maze2) {
	double diff;
	cout << endl << "BFS MAZE SOLVER WITH CPU" << endl;
	double start = seconds();
	CPU_bfs_maze_solver(maze1, side, side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze1,side,side);
	cout << endl << "BFS MAZE SOLVER WITH GPU" << endl;
	start = seconds();
	GPU_bfs_maze_solver(maze2,side,side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze2,side,side);
	hipDeviceReset();
}

void division(int side, int *maze1, int *maze2) {
	double diff;
	cout << endl << "DIVISION MAZE GENERATOR WITH CPU" << endl;
	double start = seconds();
//	CPU_division_maze_generator(maze1, side, side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze1,side,side);
	cout << endl << "DIVISION MAZE GENERATOR WITH GPU" << endl;
	start = seconds();
//	GPU_division_maze_generator(maze2,side,side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze2,side,side);
	hipDeviceReset();
}

void recursive(int side, int *maze1, int *maze2) {
	double diff;
	cout << endl << "RECURSIVE MAZE SOLVER WITH CPU" << endl;
	double start = seconds();
	CPU_recursive_maze_solver(maze1, side, side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze1,side,side);
	cout << endl << "RECURSIVE MAZE SOLVER WITH GPU" << endl;
	start = seconds();
	GPU_recursive_maze_solver(maze2,side,side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze2,side,side);
	hipDeviceReset();
}

void kruskal(int side, int *maze1, int *maze2) { //solo dispari
	double diff;
	cout << endl << "KRUSKAL MAZE GENERATOR WITH CPU" << endl;
	double start = seconds();
	CPU_kruskal_maze_generator(maze1, side, side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze1,side,side);
	cout << endl << "KRUSKAL MAZE GENERATOR WITH GPU" << endl;
	start = seconds();
	GPU_kruskal_maze_generator(maze2,side,side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze2,side,side);
	hipDeviceReset();
}

void wall_follower(int side, int startP, int endP, int *maze1, int *maze2) {
	double diff;
	cout << endl << "WALL FOLLOWER MAZE SOLVER WITH CPU" << endl;
	double start = seconds();
	CPU_wall_follower_maze_solver(maze1, startP, endP, side, side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze1,side,side);
	cout << endl << "WALL FOLLOWER MAZE SOLVER WITH GPU" << endl;
	start = seconds();
	GPU_wall_follower_maze_solver(maze2,startP, endP, side,side);
	diff = seconds()-start;
	cout << endl << "The time of computation is " << diff << " seconds" << endl;
	PrintMaze(maze2,side,side);
	hipDeviceReset();
}

//int main(){
//	int side = 1;
//	for (int i = 0; i < NUMBER_OF_TEST; i++) {
//		// setted size of the matrix
//		side *= 10;
//		int dim = side * side;
//		int maze1[dim];
//		int maze2[dim];
//
//		//DFS
//		dfs(side, maze1, maze2);
//		// Cellular automata
//		cellular(side-1, maze1,maze2);
//
//		// Backtracker
//		backtracker(side, maze1, maze2);
//		// Wall follower
//		wall_follower(side, 1, dim-2, maze1,maze2);
//
//
//		// Division
//		division(side, maze1, maze2);
//		// BFS
//		bfs(side, maze1,maze2);
//
//		// Kruskal
//		kruskal(side-1, maze1, maze2);
//		// Recursive
//		recursive(side-1, maze1,maze2);
//
//	}
//	return 0;
//}

int main() {
#define side 10
	//int side = 10;
	int dim = side * side;
	int maze[dim];
	GPU_division_maze_generator(maze,side,side);
	PrintMaze(maze,side,side);
	return 0;
}
