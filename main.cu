#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include "Header/CPU_time.h"

#include "Header/common.h"
#include "Header/backtracker_maze_generator.h"
#include "Header/cellular_automata_solver.h"
#include "Header/dfs_maze_generator.h"
#include "Header/wall_follower_maze_solver.h"
#include "Header/division_maze_generator.h"
#include "Header/kruskal_maze_generator.h"
#include "Header/bfs_maze_solver.h"

#define DIM 81
#define SIDE 9

int main(){
	//generate
	int maze[DIM];
	//CPU_kruskal_maze_generator(maze, SIDE, SIDE);
	CPU_backtracker_maze_generator(maze, SIDE, SIDE);
	maze[1] = OBJECTIVE;
	maze[79] = OBJECTIVE;
	PrintMaze(maze,SIDE,SIDE);
	CPU_bfs_maze_solver(maze,SIDE,SIDE);
	return 0;
}
