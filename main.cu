#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "Header/common.h"
#include "Header/backtracker_maze_generator.h"
#include "Header/cellular_automata_solver.h"
#include "Header/dfs_maze_generator.h"
#include "Header/wall_follower_maze_solver.h"

#define DIM 25
#define SIDE 5

int main(){
	//generate
	int maze[DIM];
    CPU_dfs_maze_generator(maze, SIDE + 1, SIDE + 1);

    //create start and end
//    int start = 0;
//    int end = DIM - 1;
//    maze[start] = OBJECTIVE;
//    maze[end] = OBJECTIVE;

    PrintMaze(maze,SIDE,SIDE);
    //solve the maze
    CPU_wall_follower_maze_solver(maze,start,end,SIDE,SIDE);
    PrintMaze(maze,SIDE,SIDE);
	return 0;
}
