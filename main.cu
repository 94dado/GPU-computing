#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include "Header/CPU_time.h"

#include "Header/common.h"
#include "Header/backtracker_maze_generator.h"
#include "Header/cellular_automata_solver.h"
#include "Header/dfs_maze_generator.h"
#include "Header/wall_follower_maze_solver.h"
#include "Header/division_maze_generator.h"
#include "Header/kruskal_maze_generator.h"
#include "Header/bfs_maze_solver.h"
#include "Header/recursive_maze_solver.h"

#define DIM 25
#define SIDE 5

int main(){
	//generate
	int maze[DIM];
	CPU_backtracker_maze_generator(maze, SIDE, SIDE);
	maze[1] = OBJECTIVE;
	maze[DIM-2] = OBJECTIVE;
	PrintMaze(maze,SIDE,SIDE);
	CPU_recursive_maze_solver(maze,SIDE,SIDE);
	hipDeviceReset();
	return 0;
}
