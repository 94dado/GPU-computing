#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "Header/common.h"

int check_neighbour_open(int *maze, int length, int row_length, int pos, int neighbour){
	if(neighbour >= 0 && neighbour < length){
		if(pos % row_length == row_length -1 && neighbour == pos + 1)
			return 0;
		if(pos % row_length == 0 && neighbour == pos - 1)
			return 0;
		if(maze[neighbour] == OPEN || maze[neighbour] == OBJECTIVE){
			return 1;
		}else{
			return 0;
		}
	}
	return 0;
}

void CPU_cellular_automata_solver(int *maze, int length, int row_length){
	bool again = true;
	int i;
	while(again){
		//risetto il booleano
		again = false;
		//per ogni cella
		for(i=0; i < length; i++){
			if(maze[i] != WALL){
				//controllo le celle vicine
				int count = 0;
				//su
				count += check_neighbour_open(maze, length, row_length, i, i - row_length);
				//giu
				count += check_neighbour_open(maze, length, row_length, i, i + row_length);
				//dx
				count += check_neighbour_open(maze, length, row_length, i, i + 1);
				//sx
				count += check_neighbour_open(maze, length, row_length, i, i - 1);
				//se ho solo 1 vicino open
				if(count == 1 && maze[i] == OPEN){
					maze[i] = WALL;
					again = true;
				}
			}
		}
	}
}

__device__ int DEVICE_check_neighbour_open(int *maze, int length, int row_length, int pos, int neighbour){
	if(neighbour >= 0 && neighbour < length){
			if(pos % row_length == row_length -1 && neighbour == pos + 1)
				return 0;
			if(pos % row_length == 0 && neighbour == pos - 1)
				return 0;
			if(maze[neighbour] == OPEN || maze[neighbour] == OBJECTIVE){
				return 1;
			}else{
				return 0;
			}
		}else{
			return 0;
		}
}

__global__ void GPU_check_neighbour_open(int *maze, int length, int row_length, bool *again){
	int i = blockIdx.x * row_length + threadIdx.x;
	//controllo le celle vicine
	int count = 0;
	//su
	count += DEVICE_check_neighbour_open(maze, length, row_length, i,  i - row_length);
	//giu
	count += DEVICE_check_neighbour_open(maze, length, row_length, i, i + row_length);
	//dx
	count += DEVICE_check_neighbour_open(maze, length, row_length, i, i + 1);
	//sx
	count += DEVICE_check_neighbour_open(maze, length, row_length, i, i - 1);
	//attendo che tutti abbiano fatto
	__syncthreads();
	if(count == 1 && maze[i] == OPEN){
		maze[i] = WALL;
		*again = true;
	}

}

void GPU_cellular_automata_solver(int *maze, int length, int row_length){
	bool again = true;
	int *dev_maze;
	bool *dev_again;
	//copio su device
	hipMalloc(&dev_maze, sizeof(int) * length);
	hipMalloc(&dev_again, sizeof(bool));

	hipMemcpy(dev_maze,maze,sizeof(int)*length,hipMemcpyHostToDevice);
	while(again){
		//cambio valore booleano per fermarmi se servira'
		again = false;
		hipMemcpy(dev_again, &again, sizeof(bool), hipMemcpyHostToDevice);
		GPU_check_neighbour_open<<<length/row_length, row_length>>>(dev_maze, length, row_length, dev_again);
		//attendo
		hipDeviceSynchronize();
		//copio su host
		hipMemcpy(&again, dev_again, sizeof(bool), hipMemcpyDeviceToHost);
	}
	//terminata esecuzione su gpu. copio risultato
	hipMemcpy(maze, dev_maze, sizeof(int) * length, hipMemcpyDeviceToHost);
	//dovrebbe aver finito
}
