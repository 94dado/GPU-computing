
#include <hip/hip_runtime.h>
#include <iostream>
#include <queue>
#include <vector>

using namespace std;

// the start and the end of the path
int startPath[2];
int endPath[2];

// queue NodeStruct used in BFS
struct NodeStruct {
    // (x, y) represents matrix cell cordinates
    // dist represent its minimum distance from the source
    int x, y;
    NodeStruct *parent;
    bool isNotWall;
};

// Below arrays details all 4 possible movements from a cell
int row[] = { -1, 0, 0, 1 };
int col[] = { 0, -1, 1, 0 };

// Function to check if it is possible to go to position (row, col)
// from current position. The function returns false if the cell
// not a valid position or has value 0 or it is already visited
bool isValid(NodeStruct *mat, bool *visited, int row, int col, int width, int height) {
    return (row >= 0) && (row < width) && (col >= 0) && (col < height) && mat[row * width + col].isNotWall && !visited[row * width + col];
}

// print the path
void PrintNodeMaze(NodeStruct *array, int width, int height){
    int i,j;
    for(i = 0; i < height; i++){
        for(j = 0; j < width; j++){
            if ((array[i*width + j].x == startPath[0] && array[i*width + j].y == startPath[1]) || (array[i*width + j].x == endPath[0] && array[i*width + j].y == endPath[1])) {
                cout << 2 << " ";
            }
            else {
                cout << array[i*width + j].isNotWall << " ";
            }
        }
        cout << endl;
    }
    cout << endl;
}

// Search the shortest path by parent NodeStruct
void ReachPath(NodeStruct *matrix, int width, int height) {
    for (int i = 0; i < width * height; i++) {
        matrix[i].isNotWall = false;
    }
    // the last is the end
    NodeStruct *currentNodeStruct = &(matrix[endPath[0] * width + endPath[1]]);
    // the first is the start
    NodeStruct *startNodeStruct = &(matrix[startPath[0] * width + startPath[1]]);
    bool isReach = false;
    while (!isReach) {
        matrix[currentNodeStruct->x * width + currentNodeStruct->y].isNotWall = true;
        currentNodeStruct = currentNodeStruct->parent;
        if (currentNodeStruct->x == startNodeStruct->x && currentNodeStruct->y == startNodeStruct->y) {
            isReach = true;
        }
    }
}

// it find start and end for the parameter of the BFS algorithm
void FromCoordToNodeStruct(NodeStruct *matrix, int *mat, int width, int height) {
    bool findStart = false;
    for(int i = 0; i < width; i++){
        for(int j = 0; j < height; j++){
            if (mat[i*width + j] == 1) {
                // is not a wall
                matrix[i * width + j] = {i, j, NULL, true};
            }
            // is the start
            else if (mat[i*width + j] == 2 && !findStart) {
                matrix[i * width + j] = {i, j, NULL, true};
                startPath[0] = i;
                startPath[1] = j;
                findStart = true;
            }
            // is the end
            else if (mat[i*width + j] == 2 && findStart) {
                matrix[i * width + j] = {i, j, NULL, true};
                endPath[0] = i;
                endPath[1] = j;
            }
            else {
                matrix[i * width + j] = {i, j, NULL, false};
            }
        }
    }
}

// Find Shortest Possible Route in a matrix mat from source
// cell (i, j) to destination cell (x, y)
void CPU_bfs_maze_solver(int *mat, int width, int height) {
    // pass to NodeStruct coordinates
    NodeStruct matrix[width * height];
    FromCoordToNodeStruct(matrix, mat, width, height);
    // construct a matrix to keep track of visited cells
    bool visited[width * height];

    // initially all cells are unvisited
    for (int k = 0; k < width * height; k++) {
        visited[k] = false;
    }

    // create an empty queue
    queue<NodeStruct> q;

    // mark source cell as visited and enqueue the source NodeStruct
    visited[startPath[0] * width + startPath[1]] = true;
    q.push(matrix[startPath[0] * width + startPath[1]]);

    // run till queue is not empty
    while (!q.empty()) {
        // pop front NodeStruct from queue and process it
        NodeStruct NodeStruct = q.front();
        q.pop();

        // (i, j) represents current cell and dist stores its
        // minimum distance from the source
        int i = NodeStruct.x, j = NodeStruct.y;

        // if destination is found, update min_dist and stop
        if (i == endPath[0] && j == endPath[1]) {
            ReachPath(matrix, width, height);
//            PrintNodeMaze(matrix, width, height);
            break;
        }

        // check for all 4 possible movements from current cell
        // and enqueue each valid movement into the queue
        for (int k = 0; k < 4; k++) {
            // check if it is possible to go to position
            // (i + row[k], j + col[k]) from current position
            if (isValid(matrix, visited, i + row[k], j + col[k], width, height)) {
                // mark next cell as visited and enqueue it
                visited[(i + row[k]) * width + (j + col[k])] = true;
                q.push({ i + row[k], j + col[k], &matrix[i * width + j], matrix[i * width + j].isNotWall });
                matrix[(i + row[k]) * width + (j + col[k])].parent = &matrix[i * width + j];
            }
        }
    }

//    if (min_dist != INT_MAX)
//        cout << "The shortest path from source to destination "
//        "has length " << min_dist << endl;
//    else
//        cout << "Destination can't be reached from given source" << endl;
}

__global__ void GPU_FromCoordToNodeStruct(NodeStruct *matrix, int *mat, int width, int height, int i, int *startPath, int *endPath){
	bool findStart = false;
	int j = blockIdx.x * width + threadIdx.x;
	if (mat[i*width + j] == 1) {
		// is not a wall
		matrix[i * width + j] = {i, j, NULL, true};
	}
	// is the start
	else if (mat[i*width + j] == 2 && !findStart) {
		matrix[i * width + j] = {i, j, NULL, true};
		startPath[0] = i;
		startPath[1] = j;
		findStart = true;
	}
	// is the end
	else if (mat[i*width + j] == 2 && findStart) {
		matrix[i * width + j] = {i, j, NULL, true};
		endPath[0] = i;
		endPath[1] = j;
	}
	else {
		matrix[i * width + j] = {i, j, NULL, false};
	}
}

__global__ void setupVisited(bool *visited, int width, int height){
	int k = blockIdx.x * width + threadIdx.x;
	visited[k] = false;
}

void GPU_bfs_maze_solver(int *mat, int width, int height){
	// pass to NodeStruct coordinates
	    NodeStruct matrix[width * height];

	    //cuda variable
	    NodeStruct *dev_matrix;
	    int *dev_startPath, *dev_endPath;
	    //memory allocation
	    hipMalloc(&dev_matrix, sizeof(NodeStruct) * width * height);
	    hipMalloc(&dev_startPath, sizeof(int) * 2);
	    hipMalloc(&dev_endPath, sizeof(int) * 2);
	    //copy data on GPU
	    hipMemcpy(&dev_matrix, matrix, sizeof(NodeStruct) * width * height, hipMemcpyHostToDevice);

	    for(int i = 0; i < width; i++){
	    	GPU_FromCoordToNodeStruct<<<width/32, 32>>>(matrix, mat, width, height, i, dev_startPath, dev_endPath);
	    }
	    hipDeviceSynchronize();
	    //get back all the data
	    hipMemcpy(matrix, dev_matrix, sizeof(NodeStruct) * width * height, hipMemcpyDeviceToHost);
	    hipMemcpy(startPath, dev_startPath, sizeof(int) * 2, hipMemcpyDeviceToHost);
	    hipMemcpy(endPath, dev_endPath, sizeof(int) * 2, hipMemcpyDeviceToHost);

	    // construct a matrix to keep track of visited cells
	    bool visited[width * height];

	    bool *dev_visited;
	    hipMalloc(&dev_visited, sizeof(bool) * width * height);
	    // initially all cells are unvisited
	   setupVisited<<<width, height>>>(dev_visited, width, height);
	   hipDeviceSynchronize();
	   hipMemcpy(visited, dev_visited, sizeof(bool) * width * height, hipMemcpyDeviceToHost);
	    // create an empty queue
	    queue<NodeStruct> q;

	    // mark source cell as visited and enqueue the source NodeStruct
	    visited[startPath[0] * width + startPath[1]] = true;
	    q.push(matrix[startPath[0] * width + startPath[1]]);

	    // run till queue is not empty
	    while (!q.empty()) {
	        // pop front NodeStruct from queue and process it
	        NodeStruct NodeStruct = q.front();
	        q.pop();

	        // (i, j) represents current cell and dist stores its
	        // minimum distance from the source
	        int i = NodeStruct.x, j = NodeStruct.y;

	        // if destination is found, update min_dist and stop
	        if (i == endPath[0] && j == endPath[1]) {
	            ReachPath(matrix, width, height);
	            PrintNodeMaze(matrix, width, height);
	            break;
	        }

	        // check for all 4 possible movements from current cell
	        // and enqueue each valid movement into the queue
	        for (int k = 0; k < 4; k++) {
	            // check if it is possible to go to position
	            // (i + row[k], j + col[k]) from current position
	            if (isValid(matrix, visited, i + row[k], j + col[k], width, height)) {
	                // mark next cell as visited and enqueue it
	                visited[(i + row[k]) * width + (j + col[k])] = true;
	                q.push({ i + row[k], j + col[k], &matrix[i * width + j], matrix[i * width + j].isNotWall });
	                matrix[(i + row[k]) * width + (j + col[k])].parent = &matrix[i * width + j];
	            }
	        }
	    }
}

